#include "hip/hip_runtime.h"
//#define SAVE_ME_FROM_FLTNUMMAT

//#define CERR
#define PI_4I 0.079577471F
//#define PI3_4I 0.238732413F
#include <cutil.h>
//#include <cutil_inline.h>
#include "../p3d/upComp.h"
#include "../p3d/dnComp.h"
#include "../p3d/point3d.h"//dont remove this
#include "gpu_setup.h"

#include <cstdio>

//#include <iostream>

#define BLOCK_HEIGHT 64

__constant__ float3 sampos[320];	//undefined for everything greater than 295 for 6, greater than 191 for 4

__constant__ float3 samposDn[152];	//undefined for everything greater than 151 for 6 and 55 for 4

__global__ void up_kernel(float *src_dp,float *trgVal_dp,float *trgCtr_dp,float *trgRad_dp,int *srcBox_dp,int numSrcBox) {
	__shared__ float4 s_sh[BLOCK_HEIGHT];

	int uniqueBlockId=blockIdx.y * gridDim.x + blockIdx.x;
	if(uniqueBlockId<numSrcBox) {
		float3 trgCtr;
		float trgRad;
	//	float3 samp[5];
		float3 trg[5];
		float dX_reg;
		float dY_reg;
		float dZ_reg;
		int2 src=((int2*)srcBox_dp)[uniqueBlockId];	//x has start, y has size
		src.x+=threadIdx.x;

		trgCtr=((float3*)trgCtr_dp)[uniqueBlockId];
		trgRad=trgRad_dp[uniqueBlockId];

		//construct the trg

		trg[0].x=trgCtr.x+trgRad*sampos[4*threadIdx.x].x;
		trg[0].y=trgCtr.y+trgRad*sampos[4*threadIdx.x].y;
		trg[0].z=trgCtr.z+trgRad*sampos[4*threadIdx.x].z;
		trg[1].x=trgCtr.x+trgRad*sampos[4*threadIdx.x+1].x;
		trg[1].y=trgCtr.y+trgRad*sampos[4*threadIdx.x+1].y;
		trg[1].z=trgCtr.z+trgRad*sampos[4*threadIdx.x+1].z;
		trg[2].x=trgCtr.x+trgRad*sampos[4*threadIdx.x+2].x;
		trg[2].y=trgCtr.y+trgRad*sampos[4*threadIdx.x+2].y;
		trg[2].z=trgCtr.z+trgRad*sampos[4*threadIdx.x+2].z;
		trg[3].x=trgCtr.x+trgRad*sampos[4*threadIdx.x+3].x;
		trg[3].y=trgCtr.y+trgRad*sampos[4*threadIdx.x+3].y;
		trg[3].z=trgCtr.z+trgRad*sampos[4*threadIdx.x+3].z;
		trg[4].x=trgCtr.x+trgRad*sampos[256+threadIdx.x].x;
		trg[4].y=trgCtr.y+trgRad*sampos[256+threadIdx.x].y;
		trg[4].z=trgCtr.z+trgRad*sampos[256+threadIdx.x].z;

	//	int numSrc=srcBoxSize[uniqueBlockId];

		float4 tv=make_float4(0.0F,0.0F,0.0F,0.0F);
		float tve=0.0F;






		int num_chunk_loop=src.y/BLOCK_HEIGHT;
		for(int chunk=0;chunk<num_chunk_loop;chunk++) {
			__syncthreads();
			s_sh[threadIdx.x]=((float4*)src_dp)[src.x];
			__syncthreads();

			src.x+=BLOCK_HEIGHT;

			for(int s=0;s<BLOCK_HEIGHT;s++) {
				dX_reg=s_sh[s].x-trg[0].x;
				dY_reg=s_sh[s].y-trg[0].y;
				dZ_reg=s_sh[s].z-trg[0].z;

				dX_reg*=dX_reg;
				dY_reg*=dY_reg;
				dZ_reg*=dZ_reg;

				dX_reg += dY_reg+dZ_reg;

				dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
				tv.x+=dX_reg*s_sh[s].w;
				///////////////////////////////
				dX_reg=s_sh[s].x-trg[1].x;
				dY_reg=s_sh[s].y-trg[1].y;
				dZ_reg=s_sh[s].z-trg[1].z;

				dX_reg*=dX_reg;
				dY_reg*=dY_reg;
				dZ_reg*=dZ_reg;

				dX_reg += dY_reg+dZ_reg;

				dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
				tv.y+=dX_reg*s_sh[s].w;
				///////////////////////////////
				dX_reg=s_sh[s].x-trg[2].x;
				dY_reg=s_sh[s].y-trg[2].y;
				dZ_reg=s_sh[s].z-trg[2].z;

				dX_reg*=dX_reg;
				dY_reg*=dY_reg;
				dZ_reg*=dZ_reg;

				dX_reg += dY_reg+dZ_reg;

				dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
				tv.z+=dX_reg*s_sh[s].w;
				///////////////////////////////
				dX_reg=s_sh[s].x-trg[3].x;
				dY_reg=s_sh[s].y-trg[3].y;
				dZ_reg=s_sh[s].z-trg[3].z;

				dX_reg*=dX_reg;
				dY_reg*=dY_reg;
				dZ_reg*=dZ_reg;

				dX_reg += dY_reg+dZ_reg;

				dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
				tv.w+=dX_reg*s_sh[s].w;
				///////////////////////////////
				dX_reg=s_sh[s].x-trg[4].x;
				dY_reg=s_sh[s].y-trg[4].y;
				dZ_reg=s_sh[s].z-trg[4].z;

				dX_reg*=dX_reg;
				dY_reg*=dY_reg;
				dZ_reg*=dZ_reg;

				dX_reg += dY_reg+dZ_reg;

				dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
				tve+=dX_reg*s_sh[s].w;
				///////////////////////////////
			}

		}	//end num chunk loop
		__syncthreads();
		s_sh[threadIdx.x]=((float4*)src_dp)[src.x];
		__syncthreads();
		for(int s=0;s<src.y%BLOCK_HEIGHT;s++) {
			dX_reg=s_sh[s].x-trg[0].x;
			dY_reg=s_sh[s].y-trg[0].y;
			dZ_reg=s_sh[s].z-trg[0].z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
			tv.x+=dX_reg*s_sh[s].w;
			///////////////////////////////
			dX_reg=s_sh[s].x-trg[1].x;
			dY_reg=s_sh[s].y-trg[1].y;
			dZ_reg=s_sh[s].z-trg[1].z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
			tv.y+=dX_reg*s_sh[s].w;
			///////////////////////////////
			dX_reg=s_sh[s].x-trg[2].x;
			dY_reg=s_sh[s].y-trg[2].y;
			dZ_reg=s_sh[s].z-trg[2].z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
			tv.z+=dX_reg*s_sh[s].w;
			///////////////////////////////
			dX_reg=s_sh[s].x-trg[3].x;
			dY_reg=s_sh[s].y-trg[3].y;
			dZ_reg=s_sh[s].z-trg[3].z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
			tv.w+=dX_reg*s_sh[s].w;
			///////////////////////////////
			dX_reg=s_sh[s].x-trg[4].x;
			dY_reg=s_sh[s].y-trg[4].y;
			dZ_reg=s_sh[s].z-trg[4].z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
			tve+=dX_reg*s_sh[s].w;
			///////////////////////////////
		}	//end residual loop

		//write back
		tv.x*=PI_4I;
		tv.y*=PI_4I;
		tv.z*=PI_4I;
		tv.w*=PI_4I;
	//	tv.x=(float)trgCtr;
	//	tv.y=tv.z=tv.w=0.0F;
		((float4*)trgVal_dp)[uniqueBlockId*74+threadIdx.x]=tv;
		if(threadIdx.x<40)
			trgVal_dp[uniqueBlockId*296+256+threadIdx.x]=tve*PI_4I;
	}

}

__global__ void up_kernel_4(float *src_dp,float *trgVal_dp,float *trgCtr_dp,float *trgRad_dp,int *srcBox_dp,int numSrcBox) {
	__shared__ float4 s_sh[BLOCK_HEIGHT];

	int uniqueBlockId=blockIdx.y * gridDim.x + blockIdx.x;
	if(uniqueBlockId<numSrcBox) {
		float3 trgCtr;
		float trgRad;
	//	float3 samp[5];
		float3 trg[3];
		float dX_reg;
		float dY_reg;
		float dZ_reg;
		int2 src=((int2*)srcBox_dp)[uniqueBlockId];	//x has start, y has size
		src.x+=threadIdx.x;

		trgCtr=((float3*)trgCtr_dp)[uniqueBlockId];
		trgRad=trgRad_dp[uniqueBlockId];

		//construct the trg

		trg[0].x=trgCtr.x+trgRad*sampos[2*threadIdx.x].x;
		trg[0].y=trgCtr.y+trgRad*sampos[2*threadIdx.x].y;
		trg[0].z=trgCtr.z+trgRad*sampos[2*threadIdx.x].z;
		trg[1].x=trgCtr.x+trgRad*sampos[2*threadIdx.x+1].x;
		trg[1].y=trgCtr.y+trgRad*sampos[2*threadIdx.x+1].y;
		trg[1].z=trgCtr.z+trgRad*sampos[2*threadIdx.x+1].z;
		trg[2].x=trgCtr.x+trgRad*sampos[128+threadIdx.x].x;		//128 is blockheight*(trg2fetch-1)
		trg[2].y=trgCtr.y+trgRad*sampos[128+threadIdx.x].y;
		trg[2].z=trgCtr.z+trgRad*sampos[128+threadIdx.x].z;

	//	int numSrc=srcBoxSize[uniqueBlockId];

		float2 tv=make_float2(0.0F,0.0F);					//can be converted into a generic array.. not too big
		float tve=0.0F;






		int num_chunk_loop=src.y/BLOCK_HEIGHT;
		for(int chunk=0;chunk<num_chunk_loop;chunk++) {
			__syncthreads();
			s_sh[threadIdx.x]=((float4*)src_dp)[src.x];
			__syncthreads();

			src.x+=BLOCK_HEIGHT;

			for(int s=0;s<BLOCK_HEIGHT;s++) {
				dX_reg=s_sh[s].x-trg[0].x;
				dY_reg=s_sh[s].y-trg[0].y;
				dZ_reg=s_sh[s].z-trg[0].z;

				dX_reg*=dX_reg;
				dY_reg*=dY_reg;
				dZ_reg*=dZ_reg;

				dX_reg += dY_reg+dZ_reg;

				dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
				tv.x+=dX_reg*s_sh[s].w;
				///////////////////////////////
				dX_reg=s_sh[s].x-trg[1].x;
				dY_reg=s_sh[s].y-trg[1].y;
				dZ_reg=s_sh[s].z-trg[1].z;

				dX_reg*=dX_reg;
				dY_reg*=dY_reg;
				dZ_reg*=dZ_reg;

				dX_reg += dY_reg+dZ_reg;

				dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
				tv.y+=dX_reg*s_sh[s].w;
				///////////////////////////////
				dX_reg=s_sh[s].x-trg[2].x;
				dY_reg=s_sh[s].y-trg[2].y;
				dZ_reg=s_sh[s].z-trg[2].z;

				dX_reg*=dX_reg;
				dY_reg*=dY_reg;
				dZ_reg*=dZ_reg;

				dX_reg += dY_reg+dZ_reg;

				dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
				tve+=dX_reg*s_sh[s].w;
			}
		}	//end num chunk loop
		__syncthreads();
		s_sh[threadIdx.x]=((float4*)src_dp)[src.x];
		__syncthreads();
		for(int s=0;s<src.y%BLOCK_HEIGHT;s++) {
			dX_reg=s_sh[s].x-trg[0].x;
			dY_reg=s_sh[s].y-trg[0].y;
			dZ_reg=s_sh[s].z-trg[0].z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
			tv.x+=dX_reg*s_sh[s].w;
			///////////////////////////////
			dX_reg=s_sh[s].x-trg[1].x;
			dY_reg=s_sh[s].y-trg[1].y;
			dZ_reg=s_sh[s].z-trg[1].z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
			tv.y+=dX_reg*s_sh[s].w;
			///////////////////////////////
			dX_reg=s_sh[s].x-trg[2].x;
			dY_reg=s_sh[s].y-trg[2].y;
			dZ_reg=s_sh[s].z-trg[2].z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);
			tve+=dX_reg*s_sh[s].w;
			///////////////////////////////
		}	//end residual loop

		//write back
		tv.x*=PI_4I;
		tv.y*=PI_4I;
	//	tv.x=(float)trgCtr;
	//	tv.y=tv.z=tv.w=0.0F;
		((float2*)(trgVal_dp+uniqueBlockId*152))[threadIdx.x]=tv;	//in generic, float3 writes will be unrolled into multiple writes
		if(threadIdx.x<24)
			trgVal_dp[uniqueBlockId*152+128+threadIdx.x]=tve*PI_4I;
	}

}


void unmake_ds_up(float *trgValE,upComp_t *UpC) {
	int t=0;
	for(int i=0;i<UpC->numSrcBox;i++) {
		for(int j=0;j<UpC->trgDim;j++) {
//			assert(UpC->trgVal[i]!=NULL);

			if(UpC->trgVal[i]!=NULL)
				UpC->trgVal[i][j]=trgValE[t];
			t++;
//			cout<<i<<","<<j<<endl;
//			cout<<trgValE[t-1]<<endl;
		}
	}
}


void make_ds_up(int *srcBox,upComp_t *UpC) {	//TODO
	int start=0;
	int t=0;
	int size;
	for(int i=0;i<UpC->numSrcBox;i++) {
		srcBox[t++]=start;
		size=UpC->srcBoxSize[i];
		srcBox[t++]=size;
		start+=size;
	}
}

void gpu_up(upComp_t *UpC) {
  GPU_MSG ("Upward computation");
  if (!UpC || !UpC->numSrcBox) { GPU_MSG ("==> No source boxes; skipping..."); return; }
  //	hipSetDevice(0);
//	unsigned int timer;
//	float ms;
//	cutCreateTimer(&timer);

	float *src_dp,*trgVal_dp,*trgCtr_dp,*trgRad_dp;
	int *srcBox_dp;

	float trgValE[UpC->trgDim*UpC->numSrcBox];
	int srcBox[2*UpC->numSrcBox];

	make_ds_up(srcBox,UpC);

	src_dp = gpu_calloc_float ((UpC->numSrc + BLOCK_HEIGHT) * (UpC->dim+1));
	trgCtr_dp = gpu_calloc_float (UpC->numSrcBox*3);
	trgRad_dp = gpu_calloc_float (UpC->numSrcBox);
	srcBox_dp = gpu_calloc_int (UpC->numSrcBox*2);
	trgVal_dp = gpu_calloc_float (UpC->trgDim*UpC->numSrcBox);

	gpu_copy_cpu2gpu_float (src_dp, UpC->src_, UpC->numSrc * (UpC->dim+1));
	gpu_copy_cpu2gpu_float (trgCtr_dp, UpC->trgCtr, UpC->numSrcBox*3);
	gpu_copy_cpu2gpu_float (trgRad_dp, UpC->trgRad, UpC->numSrcBox);
	gpu_copy_cpu2gpu_int (srcBox_dp, srcBox, UpC->numSrcBox*2);

	hipMemcpyToSymbol(HIP_SYMBOL(sampos),UpC->samPosF/*samp*/,sizeof(float)*UpC->trgDim*3); GPU_CE;
	int GRID_WIDTH=(int)ceil((float)UpC->numSrcBox/65535.0F);
	int GRID_HEIGHT=(int)ceil((float)UpC->numSrcBox/(float)GRID_WIDTH);
	dim3 GridDim(GRID_HEIGHT, GRID_WIDTH);
//	cout<<"Width: "<<GRID_WIDTH<<" HEIGHT: "<<GRID_HEIGHT<<endl;
	if(UpC->trgDim==296) {
		up_kernel<<<GridDim,BLOCK_HEIGHT>>>(src_dp,trgVal_dp,trgCtr_dp,trgRad_dp,srcBox_dp,UpC->numSrcBox);
	}
	else if(UpC->trgDim==152) {
		up_kernel_4<<<GridDim,BLOCK_HEIGHT>>>(src_dp,trgVal_dp,trgCtr_dp,trgRad_dp,srcBox_dp,UpC->numSrcBox);
	}
	else
	  GPU_MSG ("Upward computations not implemented for this accuracy"); //Exit the process?
		//also, a generic call can be put here
	GPU_CE;

	gpu_copy_gpu2cpu_float (trgValE, trgVal_dp, UpC->trgDim*UpC->numSrcBox);
//	CUT_SAFE_CALL(cutStopTimer(timer));
//	ms = cutGetTimerValue(timer);
//	cout<<"Up kernel: "<<ms<<"ms"<<endl;
	unmake_ds_up(trgValE,UpC);	//FIXME: copies the gpu output into the 2d array used by the interface... make the interface use a 1d array

	hipFree(src_dp); GPU_CE;
	hipFree(trgCtr_dp); GPU_CE;
	hipFree(trgRad_dp); GPU_CE;
	hipFree(srcBox_dp); GPU_CE;
	hipFree(trgVal_dp); GPU_CE;
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void make_ds_down(int *trgBox,dnComp_t *DnC) {
	int tt=0;
	int tot=0;
	for(int i=0;i<DnC->numTrgBox;i++) {
		int rem=DnC->trgBoxSize[i];
		while(rem>0) {
			trgBox[tt++]=tot;		//start
			int size=(rem<BLOCK_HEIGHT)?rem:BLOCK_HEIGHT;
			trgBox[tt++]=size;		//size
			trgBox[tt++]=i;			//box
			tot+=size;
			rem-=size;
		}
	}
}

void unmake_ds_down(float *trgValE,dnComp_t *DnC) {
	int t=0;
	for(int i=0;i<DnC->numTrgBox;i++) {
		for(int j=0;j<DnC->trgBoxSize[i];j++) {
			if(DnC->trgVal[i]!=NULL) {
				DnC->trgVal[i][j]=trgValE[t++];
//				cout<<DnC->trgVal[i][j]<<endl;
			}
		}
	}
}

__global__ void dn_kernel(float *trg_dp,float *trgVal_dp,float *srcCtr_dp,float *srcRad_dp,int *trgBox_dp,float *srcDen_dp,int numAugTrg) {
	__shared__ float4 s_sh[64];
	int3 trgBox;

	int uniqueBlockId=blockIdx.y * gridDim.x + blockIdx.x;
	if(uniqueBlockId<numAugTrg) {
		trgBox=((int3*)trgBox_dp)[uniqueBlockId];		//start,size,box

		float3 t_reg=((float3*)trg_dp)[trgBox.x+threadIdx.x];

		float3 srcCtr=((float3*)srcCtr_dp)[trgBox.z];
		float srcRad=srcRad_dp[trgBox.z];

		float dX_reg,dY_reg,dZ_reg;
		float tv_reg=0.0;

		//every thread computes a single src body


		s_sh[threadIdx.x].x=srcCtr.x+srcRad*samposDn[threadIdx.x].x;
		s_sh[threadIdx.x].y=srcCtr.y+srcRad*samposDn[threadIdx.x].y;
		s_sh[threadIdx.x].z=srcCtr.z+srcRad*samposDn[threadIdx.x].z;

		s_sh[threadIdx.x].w=srcDen_dp[152*trgBox.z+threadIdx.x];

		__syncthreads();
		for(int src=0;src<64;src++) {
			dX_reg=s_sh[src].x-t_reg.x;
			dY_reg=s_sh[src].y-t_reg.y;
			dZ_reg=s_sh[src].z-t_reg.z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);

			tv_reg+=dX_reg*s_sh[src].w;
		}
		__syncthreads();
		s_sh[threadIdx.x].x=srcCtr.x+srcRad*samposDn[64+threadIdx.x].x;
		s_sh[threadIdx.x].y=srcCtr.y+srcRad*samposDn[64+threadIdx.x].y;
		s_sh[threadIdx.x].z=srcCtr.z+srcRad*samposDn[64+threadIdx.x].z;

		s_sh[threadIdx.x].w=srcDen_dp[152*trgBox.z+threadIdx.x+64];

		__syncthreads();
		for(int src=0;src<64;src++) {
			dX_reg=s_sh[src].x-t_reg.x;
			dY_reg=s_sh[src].y-t_reg.y;
			dZ_reg=s_sh[src].z-t_reg.z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);

			tv_reg+=dX_reg*s_sh[src].w;
		}
		__syncthreads();
		if(threadIdx.x<24) {
			s_sh[threadIdx.x].x=srcCtr.x+srcRad*samposDn[128+threadIdx.x].x;
			s_sh[threadIdx.x].y=srcCtr.y+srcRad*samposDn[128+threadIdx.x].y;
			s_sh[threadIdx.x].z=srcCtr.z+srcRad*samposDn[128+threadIdx.x].z;

			s_sh[threadIdx.x].w=srcDen_dp[152*trgBox.z+threadIdx.x+128];
		}

		__syncthreads();
		for(int src=0;src<24;src++) {
			dX_reg=s_sh[src].x-t_reg.x;
			dY_reg=s_sh[src].y-t_reg.y;
			dZ_reg=s_sh[src].z-t_reg.z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);

			tv_reg+=dX_reg*s_sh[src].w;
		}

		if(threadIdx.x<trgBox.y)
			trgVal_dp[trgBox.x+threadIdx.x]=tv_reg*PI_4I;
//			trgVal_dp[trgBox.x+threadIdx.x]=trgBox.z;
	}//extra padding block

}

__global__ void dn_kernel_4(float *trg_dp,float *trgVal_dp,float *srcCtr_dp,float *srcRad_dp,int *trgBox_dp,float* srcDen_dp,int numAugTrg) {

	__shared__ float4 s_sh[56];
	int3 trgBox;

	int uniqueBlockId=blockIdx.y * gridDim.x + blockIdx.x;
	if(uniqueBlockId<numAugTrg) {
		trgBox=((int3*)trgBox_dp)[uniqueBlockId];		//start,size,box

		float3 t_reg=((float3*)trg_dp)[trgBox.x+threadIdx.x];

		float3 srcCtr=((float3*)srcCtr_dp)[trgBox.z];
		float srcRad=srcRad_dp[trgBox.z];

		float dX_reg,dY_reg,dZ_reg;
		float tv_reg=0.0;

		//every thread computes a single src body

		if(threadIdx.x<56) {	//no segfaults here

			s_sh[threadIdx.x].x=srcCtr.x+srcRad*samposDn[threadIdx.x].x;
			s_sh[threadIdx.x].y=srcCtr.y+srcRad*samposDn[threadIdx.x].y;
			s_sh[threadIdx.x].z=srcCtr.z+srcRad*samposDn[threadIdx.x].z;

			s_sh[threadIdx.x].w=srcDen_dp[56*trgBox.z+threadIdx.x];
		}
		__syncthreads();
		for(int src=0;src<56;src++) {
			dX_reg=s_sh[src].x-t_reg.x;

			dY_reg=s_sh[src].y-t_reg.y;

			dZ_reg=s_sh[src].z-t_reg.z;

			dX_reg*=dX_reg;
			dY_reg*=dY_reg;
			dZ_reg*=dZ_reg;

			dX_reg += dY_reg+dZ_reg;

			dX_reg = rsqrtf(dX_reg);
//@@
dX_reg = dX_reg + (dX_reg-dX_reg);
dX_reg = fmaxf(dX_reg,0.0F);

			tv_reg+=dX_reg*s_sh[src].w;
		}

		if(threadIdx.x<trgBox.y)
			trgVal_dp[trgBox.x+threadIdx.x]=tv_reg*PI_4I;
	}//extra padding block

}

int getnumAugTrg(dnComp_t *DnC) {
	int numAugTrg=0;
	for(int i=0;i<DnC->numTrgBox;i++) {
		numAugTrg+=(int)ceil((float)DnC->trgBoxSize[i]/(float)BLOCK_HEIGHT);
	}
	return numAugTrg;
}

void gpu_down(dnComp_t *DnC) {
  GPU_MSG ("Downward (combine) pass");
	int numAugTrg = getnumAugTrg(DnC);
	if (!numAugTrg) { GPU_MSG ("==> numAugTrg == 0; skipping..."); return; }
	//	hipSetDevice(0);
//	DnC->numTrgBox=75;
	float *trg_dp,*trgVal_dp,*srcCtr_dp,*srcRad_dp,*srcDen_dp;
//	int *srcBoxSize_dp,srcBoxStart_dp;
	int *trgBox_dp;	//has start and size and block
//	float trgValE[DnC->numTrg];
	float *trgValE=(float*)calloc(DnC->numTrg,sizeof(float));
	if(trgValE==NULL) GPU_MSG ("segfault imminent");
	int trgBox[3*numAugTrg];
	make_ds_down(trgBox,DnC);


	trg_dp = gpu_calloc_float ((DnC->numTrg+BLOCK_HEIGHT) * (DnC->dim));
	srcCtr_dp = gpu_calloc_float (DnC->numTrgBox*3);
	srcRad_dp = gpu_calloc_float (DnC->numTrgBox);
	trgBox_dp = gpu_calloc_int (numAugTrg*3);
	trgVal_dp = gpu_calloc_float (DnC->numTrg);
	srcDen_dp = gpu_calloc_float (DnC->numTrgBox*DnC->srcDim);

	gpu_copy_cpu2gpu_float (trg_dp, DnC->trg_, DnC->numTrg * DnC->dim);
	gpu_copy_cpu2gpu_float (srcCtr_dp, DnC->srcCtr, DnC->numTrgBox*3);
	gpu_copy_cpu2gpu_float (srcRad_dp, DnC->srcRad, DnC->numTrgBox);
	gpu_copy_cpu2gpu_int (trgBox_dp, trgBox, numAugTrg*3);
	gpu_copy_cpu2gpu_float (srcDen_dp, DnC->srcDen, DnC->numTrgBox*DnC->srcDim);
	hipMemcpyToSymbol(HIP_SYMBOL(samposDn), DnC->samPosF, sizeof(float)*DnC->srcDim*3); GPU_CE;
//	int GRID_HEIGHT=UpC->numSrcBox;
	int GRID_WIDTH=(int)ceil((float)numAugTrg/65535.0F);
	int GRID_HEIGHT=(int)ceil((float)numAugTrg/(float)GRID_WIDTH);
	dim3 GridDim(GRID_HEIGHT, GRID_WIDTH);
//	cout<<"Width: "<<GRID_WIDTH<<" HEIGHT: "<<GRID_HEIGHT<<endl;
	if(DnC->srcDim==152) {
	  dn_kernel<<<GridDim,BLOCK_HEIGHT>>>(trg_dp,trgVal_dp,srcCtr_dp,srcRad_dp,trgBox_dp,srcDen_dp,numAugTrg);
	}
	else if(DnC->srcDim==56) {
	  dn_kernel_4<<<GridDim,BLOCK_HEIGHT>>>(trg_dp,trgVal_dp,srcCtr_dp,srcRad_dp,trgBox_dp,srcDen_dp,numAugTrg);
	}
	else
	  GPU_MSG ("Downward computations not implemented for this accuracy");	//Exit the process?
		//also, a generic call can be put here
	GPU_CE;


	gpu_copy_gpu2cpu_float (trgValE, trgVal_dp, DnC->numTrg);
	unmake_ds_down(trgValE,DnC);	//FIXME: copies the gpu output into the 2d array used by the interface... make the interface use a 1d array
	free(trgValE);
	hipFree(trg_dp); GPU_CE;
	hipFree(srcCtr_dp); GPU_CE;
	hipFree(srcRad_dp); GPU_CE;
	hipFree(trgBox_dp); GPU_CE;
	hipFree(trgVal_dp); GPU_CE;
	hipFree(srcDen_dp); GPU_CE;
}
